#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

#include "image_processing.hpp"
#include "constants.hpp"

using namespace constants;

int cudaDeviceCount;
hipError_t cE;

// map each rank to a GPU
void map_rank_to_gpu(int my_rank)
{
    if ((cE = hipGetDeviceCount(&cudaDeviceCount)) != hipSuccess)
    {
        std::cerr << "Unable to determine cuda device count, error is " << hipGetErrorString(cE) << std::endl;
        return;
    }

    // Assign GPU (simple round-robin)
    int assignedGpu = my_rank % cudaDeviceCount;
    if ((cE = hipSetDevice(assignedGpu)) != hipSuccess)
    {
        std::cerr << "Unable to set cuda device, error is " << hipGetErrorString(cE) << std::endl;
        return;
    }
}

// Load an image from the input file path
cv::Mat load_image(const std::string &input_filepath)
{
    cv::Mat image = cv::imread(input_filepath, cv::IMREAD_COLOR);

    if (image.empty())
    {
        throw std::runtime_error("Could not read the image: " + input_filepath);
    }

    return image;
}

// resize the image to the desired width and height
void resize_image(cv::Mat &image, int &desired_width, int &desired_height)
{
    cv::resize(image, image, cv::Size(static_cast<int>(desired_width * SCALE_FACTOR), static_cast<int>(desired_height * SCALE_FACTOR * (static_cast<float>(CHARACTER_WIDTH) / CHARACTER_HEIGHT))));
}

// Split the image into equal parts for each rank
cv::Mat split_image(const cv::Mat &full_image, int my_rank, int num_ranks)
{
    int rows = full_image.rows;
    int rows_per_rank = rows / num_ranks;
    int remainder = rows % num_ranks;

    int start_row = my_rank * rows_per_rank + std::min(my_rank, remainder);
    int end_row = start_row + rows_per_rank - 1;
    if (my_rank < remainder)
    {
        end_row++;
    }

    // Ensure end_row does not exceed the last row of the image
    end_row = std::min(end_row, rows - 1);

    return full_image.rowRange(start_row, end_row + 1).clone();
}

// Convert the image to ASCII art
__global__ void imageToAsciiKernel(cv::cuda::PtrStepSz<uchar3> input, char *output, int width, const char *device_characters, int num_chats)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < input.rows)
    {
        uchar3 pixel = input(y, x);
        // Magic 🧙🏻🧙🏻‍♂️🧙🏻‍♀️
        float gray = 0.299f * pixel.x + 0.587f * pixel.y + 0.114f * pixel.z;
        int index = static_cast<int>(gray * (num_chats - 1) / 255);
        index = max(0, min(index, num_chats - 1));
        char ascii_char = device_characters[index];

        output[y * width + x] = ascii_char;
    }
}

// Process the image on the GPU and return the ASCII art and the ASCII image
std::pair<std::string, cv::Mat> process_image(const cv::Mat &image, bool colored_flag, int threads_x, int threads_y)
{
    cv::cuda::GpuMat d_image(image);
    size_t num_chars = image.rows * image.cols;
    char *d_ascii_art;
    hipMalloc(&d_ascii_art, sizeof(char) * num_chars);

    // Transfer CHARACTERS to device
    const char *device_characters;
    size_t char_size = CHARACTERS.size();
    hipMalloc(&device_characters, char_size);
    hipMemcpy((void *)device_characters, CHARACTERS.c_str(), char_size, hipMemcpyHostToDevice);

    dim3 threads_per_block(threads_x, threads_y);
    dim3 num_blocks((image.cols + threads_per_block.x - 1) / threads_per_block.x,
                    (image.rows + threads_per_block.y - 1) / threads_per_block.y);

    imageToAsciiKernel<<<num_blocks, threads_per_block>>>(d_image, d_ascii_art, image.cols, device_characters, char_size);
    hipDeviceSynchronize();

    // Copy the ASCII art from device to host
    std::vector<char> ascii_art(num_chars);
    hipMemcpy(&ascii_art[0], d_ascii_art, sizeof(char) * num_chars, hipMemcpyDeviceToHost);

    std::string ascii_art_str;
    // Extra space for newlines
    ascii_art_str.reserve(num_chars + image.rows);
    for (int i = 0; i < image.rows; ++i)
    {
        ascii_art_str.append(&ascii_art[i * image.cols], image.cols);
        ascii_art_str.push_back('\n');
    }

    hipFree(d_ascii_art);

    // Handling the drawing on host
    cv::Mat ascii_image = cv::Mat::zeros(image.rows * CHARACTER_HEIGHT, image.cols * CHARACTER_WIDTH, CV_8UC3);
    for (int y = 0; y < image.rows; ++y)
    {
        for (int x = 0; x < image.cols; ++x)
        {
            // Get the character at the current position
            std::string text(1, ascii_art[y * image.cols + x]);
            int pos_x = x * CHARACTER_WIDTH;
            int pox_y = y * CHARACTER_HEIGHT + CHARACTER_HEIGHT;
            cv::Scalar color = colored_flag ? image.at<cv::Vec3b>(y, x) : cv::Scalar::all(255);
            cv::putText(ascii_image, text, cv::Point(pos_x, pox_y), cv::FONT_HERSHEY_SIMPLEX, 0.5, color, 1);
        }
    }

    return std::make_pair(ascii_art_str, ascii_image);
}
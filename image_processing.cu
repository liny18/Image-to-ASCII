#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <opencv2/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudawarping.hpp>
#include <opencv2/cudaimgproc.hpp>

#include "image_processing.hpp"
#include "constants.hpp"

using namespace constants;

int cudaDeviceCount;
hipError_t cE;

// map each rank to a GPU
void map_rank_to_gpu(int my_rank)
{

    if ((cE = hipGetDeviceCount(&cudaDeviceCount)) != hipSuccess)
    {
        std::cerr << "Unable to determine cuda device count, error is " << hipGetErrorString(cE) << std::endl;
        return;
    }

    // Assign GPU (simple round-robin)
    int assignedGpu = my_rank % cudaDeviceCount;
    if ((cE = hipSetDevice(assignedGpu)) != hipSuccess)
    {
        std::cerr << "Unable to set cuda device, error is " << hipGetErrorString(cE) << std::endl;
        return;
    }
}

// Load an image from the input file path
cv::Mat load_image(const std::string &input_filepath)
{
    cv::Mat image = cv::imread(input_filepath, cv::IMREAD_COLOR);

    if (image.empty())
    {
        throw std::runtime_error("Could not read the image: " + input_filepath);
    }

    return image;
}

// resize the image to the desired width and height
void resize_image(cv::Mat &image, int &desired_width, int &desired_height)
{
    cv::cuda::GpuMat gpu_image;
    gpu_image.upload(image);

    cv::cuda::resize(gpu_image, gpu_image, cv::Size(static_cast<int>(desired_width * SCALE_FACTOR), static_cast<int>(desired_height * SCALE_FACTOR * (static_cast<float>(CHARACTER_WIDTH) / CHARACTER_HEIGHT))));

    gpu_image.download(image);
}

cv::Mat split_image(const cv::Mat &full_image, int my_rank, int num_ranks)
{
    int rows = full_image.rows;
    int rows_per_rank = rows / num_ranks;
    int remainder = rows % num_ranks;

    int start_row = my_rank * rows_per_rank + std::min(my_rank, remainder);
    int end_row = start_row + rows_per_rank - 1;
    if (my_rank < remainder)
    {
        end_row++;
    }

    // Ensure end_row does not exceed the last row of the image
    end_row = std::min(end_row, rows - 1);

    cv::cuda::GpuMat gpu_full_image;
    gpu_full_image.upload(full_image);

    cv::cuda::GpuMat gpu_image = gpu_full_image.rowRange(start_row, end_row + 1).clone();

    cv::Mat image;
    gpu_image.download(image);

    return image;
}

// process the image to get the ASCII art string and the ASCII image
std::pair<std::string, cv::Mat> process_image(const cv::Mat &image, bool colored_flag)
{
    std::string ascii_art;

    cv::Mat ascii_image(CHARACTER_HEIGHT * image.rows, CHARACTER_WIDTH * image.cols, CV_8UC3, cv::Scalar(0, 0, 0));

    for (int i = 0; i < image.rows; i++)
    {
        for (int j = 0; j < image.cols; j++)
        {
            // Magic 🧙🏻🧙🏻‍♂️🧙🏻‍♀️
            cv::Vec3b pixel = image.at<cv::Vec3b>(i, j);

            float gray = 0.299 * pixel[0] + 0.587 * pixel[1] + 0.114 * pixel[2];
            int index = static_cast<int>(gray * (CHARACTERS.size() - 1) / 255);
            char asciiChar = CHARACTERS[index];

            cv::Scalar textColor = (colored_flag) ? cv::Scalar(pixel[0], pixel[1], pixel[2]) : cv::Scalar::all(255);

            cv::putText(ascii_image, std::string(1, asciiChar), cv::Point(j * CHARACTER_WIDTH, i * CHARACTER_HEIGHT + CHARACTER_HEIGHT), cv::FONT_HERSHEY_SIMPLEX, 0.5, textColor, 1);

            ascii_art += asciiChar;
        }

        ascii_art += '\n';
    }

    return std::make_pair(ascii_art, ascii_image);
}

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>

cv::cuda::GpuMat load_image(const std::string &input_directory)
{
    cv::Mat image = cv::imread(input_directory, cv::IMREAD_COLOR);
    if (image.empty())
    {
        std::cerr << "Could not read the image." << std::endl;
        exit(EXIT_FAILURE);
    }

    cv::cuda::GpuMat gpu_image;
    gpu_image.upload(image);

    return gpu_image;
}

void resize_image(cv::cuda::GpuMat &gpu_image, bool resize_flag, int &desired_width, int &desired_height)
{
    if (resize_flag)
    {
        desired_height = (gpu_image.rows * desired_width) / gpu_image.cols;
    }
    else
    {
        desired_width = gpu_image.cols;
        desired_height = gpu_image.rows;
    }

    cv::cuda::GpuMat resized_image;
    cv::Size new_size(static_cast<int>(desired_width * SCALE_FACTOR),
                      static_cast<int>(desired_height * SCALE_FACTOR * (static_cast<float>(CHAR_WIDTH) / CHAR_HEIGHT)));

    cv::cuda::resize(gpu_image, resized_image, new_size);

    gpu_image = std::move(resized_image);
}

void process_image